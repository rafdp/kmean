
#include "kmean.h"

kmean::kmean (const int seed_,
	      const int K_,
	      const int Npoints_,
	      const int Nclusters_,
	      const int dimension_) : 
    seed           (seed_),
    K              (K_),
    Npoints        (Npoints_),
    Nclusters      (Nclusters_),
    dimension      (dimension_),
    dataD          (Npoints*Nclusters*dimension, 0.0f),
    initialLabelsD (Npoints*Nclusters),
    centroidsD     (K*dimension),
    labelsD        (Npoints*Nclusters, -1)
{
    GenerateDatasetGaussian (seed, 
		             Npoints, 
		             Nclusters,
			     dimension,
			     dataD.data().get(),
			     initialLabelsD.data().get(),
			     true);

}
kmean::~kmean () {}

void kmean::Write (std::string filenamePoints, std::string filenameCentroids)
{ 
    FILE* f_points = fopen (filenamePoints.c_str (), "w");
    if (!f_points) return;
    FILE* f_centroids = fopen (filenameCentroids.c_str (), "w");
    if (!f_centroids) {fclose (f_points); return;}

    thrust::host_vector<float> dataH (dataD);
    thrust::host_vector<int> labelsH (labelsD);
    thrust::host_vector<int> initialLabelsH (initialLabelsD);
    for (int i = 0; i < Npoints*Nclusters; i++)
    {
        for (int d = 0; d < dimension; d++)
            fprintf (f_points, "%f ", 
                     dataH[i*dimension + d]);
        fprintf (f_points, "%d %d\n", labelsH [i], initialLabelsH [i]);
    }

    fclose (f_points);
    thrust::host_vector<float> centroidsH (centroidsD);
    for (int c = 0; c < K; c++)
    {
        for (int d = 0; d < dimension; d++)
            fprintf (f_centroids, "%f ", 
                     centroidsH[c*dimension + d]);
        fprintf (f_centroids, "%d\n", c);
    }
    fclose (f_centroids);
}


void kmean::CentroidInitialization ()
{
    hiprandState* states = nullptr;
    CC(hipMalloc (&states, Npoints*dimension*sizeof (hiprandState)));
    bool singleClusterInitSetting = false;
    thrust::device_vector<float> centroidsInit (K*dimension, 0.5f);
    GenerateSingleCluster (states, 
		           seed,
		           dimension,
			   centroidsInit.data().get (),
			   0.5f/3.0f,
			   K,
			   centroidsD.data().get(),
			   &singleClusterInitSetting);
    CC(hipFree (states));
}


void kmean::Iteration ()
{
    LabelAssignmentFunctor laf (dataD.data().get(),
		                centroidsD.data().get(),
			        labelsD.data().get(),
			        dimension,
			        K);
    thrust::counting_iterator<int> pointCounter (0);
    thrust::for_each (thrust::device,
		      pointCounter,
		      pointCounter + Npoints*Nclusters,
		      laf);

    typedef IteratorSizeHelper* CustomPtr;
    
    thrust::device_vector<float> d_centroidsD (centroidsD);
    CustomPtr dataCustomPtr = reinterpret_cast<CustomPtr> (dataD.data().get());
    CustomPtr centroidsCustomPtr = reinterpret_cast<CustomPtr> (centroidsD.data().get());
    CustomPtr d_centroidsCustomPtr = reinterpret_cast<CustomPtr> (d_centroidsD.data().get());

    thrust::sort_by_key (thrust::device,
		         labelsD.begin(),
			 labelsD.end(),
			 dataCustomPtr);

    thrust::device_vector<int> keyDump (K, 0);
    thrust::reduce_by_key (thrust::device,
		           labelsD.begin(),
			   labelsD.end(),
			   dataCustomPtr,
			   keyDump.begin(),
			   d_centroidsCustomPtr,
			   thrust::equal_to<int> (),
			   thrust::plus<IteratorSizeHelper> ());
    int* keyDumpPtr = keyDump.data().get();
    thrust::device_vector<int> clusterSizes (keyDump);
    thrust::upper_bound (thrust::device,
		         labelsD.begin(),
			 labelsD.end(),
			 keyDump.begin(),
			 keyDump.end(),
			 clusterSizes.begin());
    thrust::adjacent_difference (thrust::device,
		                 clusterSizes.begin(),
				 clusterSizes.end(),
				 clusterSizes.begin());
    
    CentroidDividerFunctor cdf (d_centroidsD.data().get(),
		                clusterSizes.data().get(),
				dimension);
    thrust::for_each (pointCounter,
		      pointCounter + K,
		      cdf);
    thrust::for_each (thrust::device,
		      pointCounter,
		      pointCounter + K,
		      [centroidsCustomPtr, keyDumpPtr, d_centroidsCustomPtr]__device__ (int idx) 
		      {if (keyDumpPtr[idx] >= idx) centroidsCustomPtr[keyDumpPtr[idx]] = 
		       d_centroidsCustomPtr[idx];});
}

void kmean::Process (const int max_iter)
{
    CentroidInitialization();
    for (int i = 0; i < max_iter; i++) Iteration ();
    //printf ("About to write\n");
    //Write ("test_data.txt", "test_centroids.txt");
}

__device__
bool CentroidPointData::operator > (const CentroidPointData that) const
{
    return distance > that.distance;
}
__device__
bool CentroidPointData::operator < (const CentroidPointData that) const
{
    return distance < that.distance;
}

LabelAssignmentFunctor::LabelAssignmentFunctor (float* dataD_,
		                                float* centroidsD_,
						int* labelsD_,
						const int dimension_, 
						const int K_) : 
    dataD      (dataD_),
    centroidsD (centroidsD_),
    labelsD    (labelsD_),
    dimension  (dimension_),
    K          (K_)
{}
__device__
void LabelAssignmentFunctor::operator () (int pointIndex)
{
    thrust::minimum<CentroidPointData> minimizer;
    thrust::counting_iterator<int> centroidCounter (0);
    DistancePointToCentroidFunctor dptcf (dataD + pointIndex*dimension,
		                          centroidsD,
					  dimension);
    CentroidPointData result = {100.0f, K + 1};
    result = thrust::transform_reduce (thrust::device, 
		                       centroidCounter, 
			               centroidCounter + K,
			               dptcf,
			               result,
			               minimizer);
    labelsD[pointIndex] = result.centroid;
}
__device__
DistancePointToCentroidFunctor::DistancePointToCentroidFunctor (float* pointD_,
                                                                float* centroidsD_,
								const int dimension_) : 
    pointD     (pointD_),
    centroidsD (centroidsD_),
    dimension  (dimension_)
{}
__device__
CentroidPointData DistancePointToCentroidFunctor::operator () (int centroidIndex)
{
    float distance = 0.0f;
    float tempValue = 0.0f;
    for (int d = 0; d < dimension; d++)
    {
	tempValue = (pointD[d] - centroidsD[centroidIndex*dimension + d]);
	distance += tempValue*tempValue;
    }
    CentroidPointData cpd = {distance, centroidIndex};
    return cpd;
}

__device__
IteratorSizeHelper IteratorSizeHelper::operator+ (const IteratorSizeHelper b) const
{
    IteratorSizeHelper res = {};
    for (int d = 0; d < KMEAN_DIMENSION_DEFINED; d++)
    {
        res.data[d] = data[d] + b.data[d];
    }
    return res;
}
__device__
IteratorSizeHelper& IteratorSizeHelper::operator= (IteratorSizeHelper b) 
{
    for (int d = 0; d < KMEAN_DIMENSION_DEFINED; d++)
    {
        data[d] = b.data[d];
    }
    return *this;
}


__host__ 
CentroidDividerFunctor::CentroidDividerFunctor (float* centroidsD_,
                        int* keySizesD_,
	                const int dimension_) : 
    centroidsD (centroidsD_),
    keySizesD  (keySizesD_),
    dimension (dimension_)
{}

__device__
void CentroidDividerFunctor::operator () (int index)
{
    if (!keySizesD[index]) return;
    for (int d = 0; d < dimension; d++)
    {
        centroidsD[index*dimension + d] /= keySizesD[index];
    }

}
